#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <omp.h>
#include <cmath>
#include <iostream>
#include <cstring>

#include <stdio.h>

void lab1() {
	hipDeviceProp_t prop;

	if (hipGetDeviceProperties(&prop, 0) != hipSuccess) {
		printf("Could not get device properties");
		return;
	}

	FILE* f = fopen("results.txt", "w");

	fprintf(f, "Имя устройства %s\n", prop.name);
	fprintf(f, "Глобальная память = %d\n", prop.totalGlobalMem);
	fprintf(f, "Общая память на блок = %d\n", prop.sharedMemPerBlock);
	fprintf(f, "Количество 32-битных регистров на блок = %d\n", prop.regsPerBlock);
	fprintf(f, "Размер варпа = %d\n", prop.warpSize);
	fprintf(f, "Максимальный Pitch для функции cudaMallocpitch = %d\n ", prop.memPitch);
	fprintf(f, "Max число активных нитей в блоке = %d\n", prop.maxThreadsPerBlock);
	fprintf(f, "Max размер блока по каждому измерению = %d %d %d\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	fprintf(f, "Max размер сетки по каждому измерению = %d %d %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	fprintf(f, "Частота в килогерцах = %d\n", prop.clockRate);
	fprintf(f, "Объем константной памяти в байтах= %d\n", prop.totalConstMem);
	fprintf(f, "Compute Capability = %d.%d\n", prop.major, prop.minor);
	fprintf(f, "Выравнивание памяти для текстур = %d\n", prop.textureAlignment);
	fprintf(f, "Можно ли копировать и вычислять одновременно = %d\n", prop.deviceOverlap);
	fprintf(f, "Количество мультипроцессоров в GPU = %d\n", prop.multiProcessorCount);
	fprintf(f, "Ограничение на время выполнения ядра = %d\n", prop.kernelExecTimeoutEnabled);

	fclose(f);
}

const int n = 2;
const double tau = 0.01;
const double t0 = 0.0;
double t_max = 100.0;
double y[n] = { 0.0, 0.0 };
double yy[n], ff[n], ffk[n];

double f(double* y, double x, int i) {
	double w = 0.0, a;

	switch (i)
	{
	case 0:
		w = y[1];
		for (int i = 0; i < 10000; ++i) { a = sin((double)i); }
		break;

	case 1:
		for (int i = 0; i < 10000; ++i) { a = sin((double)i); }
		w = expf(-x * y[0]);
		break;
	}

	return w;
}

void lab2() {
	double time_begin, time_end, time_elapsed;

	time_begin = omp_get_wtime();

	for (double t = t0; t < t_max; t += tau)
	{
		for (int i = 0; i < n; ++i)
			ffk[i] = f(y, t, i);

		for (int i = 0; i < n; ++i)
			yy[i] = y[i] + tau * ffk[i];

		for (int i = 0; i < n; ++i)
			ff[i] = f(yy, t + tau, i);

		for (int i = 0; i < n; ++i)
			y[i] += tau * (ffk[i] + ff[i]) / 2.0;
	}

	time_end = omp_get_wtime();
	time_elapsed = time_end - time_begin;
	std::cout << "Time elapsed: " << time_elapsed << std::endl;

	for (int i = 0; i < n; ++i)
	{
		std::cout << "Array[" << i << "]: " << y[i] << std::endl;
	}
}




int main() {
	lab2();
	return 0;
}
